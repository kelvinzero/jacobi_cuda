#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


extern  __shared__  float sdata[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
   	extern __shared__ float s_data[];
	
	unsigned int x_global 	= blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int y_global 	= blockIdx.y * blockDim.y + threadIdx.y + 1;
	unsigned int blockWidth = blockDim.x;

	// if out of range, return
  	if( x_global >= width - 1|| y_global >= width - 1 || x_global < 1 || y_global < 1 ) 
		return;

	// set the shared data values in this area;
	
	// if left edge thread
	if(threadIdx.x == 0){
		s_data[0]	 	 = g_dataA[(y_global-1) * floatpitch + x_global - 1]; 			// NW
		s_data[blockWidth+2] 	 = g_dataA[y_global * floatpitch + x_global-1]; 			// W
		s_data[(blockWidth+2)*2] = g_dataA[(y_global+1) * floatpitch + x_global-1];			// SW
	}
	// if right edge thread
	if(threadIdx.x == blockDim.x-1 || x_global == width-2){
		s_data[threadIdx.x+2] 			 = g_dataA[(y_global-1) * floatpitch + x_global + 1];	// NE
		s_data[blockWidth+2+threadIdx.x+2] 	 = g_dataA[y_global * floatpitch + x_global+1]; 	// E
		s_data[(blockWidth+2)*2 + threadIdx.x+2] = g_dataA[(y_global+1) * floatpitch + x_global+1]; 	// SE
	}
	
	// all threads
	s_data[blockWidth+2 + threadIdx.x+1] 	 = g_dataA[y_global * floatpitch + x_global];			// CENTER
	s_data[threadIdx.x+1] 			 = g_dataA[(y_global-1) * floatpitch + x_global];		// N
	s_data[(blockWidth+2)*2 + threadIdx.x+1] = g_dataA[(y_global+1) * floatpitch + x_global];		// S
 				
	// wait for all threads to add values to shared array
	__syncthreads();

	// set the results
	g_dataB[y_global * floatpitch + x_global] = 	(
					        0.1f * s_data[threadIdx.x] +				// NW 
						0.1f * s_data[threadIdx.x+1] +				// N 
						0.1f * s_data[threadIdx.x+2] + 				// NE
						0.1f * s_data[(blockWidth+2) + threadIdx.x] +		// W
						0.2f * s_data[blockWidth+2 + threadIdx.x+1] +	 	// thisBlock
						0.1f * s_data[(blockWidth+2) + threadIdx.x+2] +		// E
						0.1f * s_data[(blockWidth+2)*2 + threadIdx.x] +		// SW
						0.1f * s_data[(blockWidth+2)*2 + threadIdx.x+1] +	// S
						0.1f * s_data[(blockWidth+2)*2 + threadIdx.x+2] 	// SE
					) * 0.95f;  	
}

